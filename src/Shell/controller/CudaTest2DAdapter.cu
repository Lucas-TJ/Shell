#include "hip/hip_runtime.h"
#include <sofa/helper/fixed_array.h>
#include <sofa/gpu/cuda/CudaCommon.h>
#include <sofa/gpu/cuda/CudaMath.h>
//#include <hip/hip_runtime.h>

#if defined(__cplusplus) && CUDA_VERSION < 2000
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

extern "C"
{
void Test2DAdapterCuda3f_computeTriangleNormal(unsigned int size, const void* x, void* tri);
void Test2DAdapterCuda3f_functionalGeom(unsigned int size, const void* x, void* tri);
void Test2DAdapterCuda3f_reduceStep(unsigned int size, void* x, void* pt, void* indices);
void Test2DAdapterCuda3f_restoreUnchanged(unsigned int size, void* x, void* pt, void* indices);
void Test2DAdapterCuda3f_smooth(unsigned int size, void* x, void* tri, void* pt, void* indices);
void Test2DAdapterCuda3f_testAcceptable(unsigned int size, void* x, const void* tri, void* pt, void* indices, float tolerance);
///// parallel
void Test2DAdapterCuda3f_prepareGradients(unsigned int size, const void* x, void* tri);
void Test2DAdapterCuda3f_smoothParallel(unsigned int size, void* x, const void* tri, void* pt);
void Test2DAdapterCuda3f_reduceStepP(unsigned int size, void* x, void* pt);
void Test2DAdapterCuda3f_testAcceptableP(unsigned int size, void* x, const void* tri, void* pt, float tolerance);
void Test2DAdapterCuda3f_restoreUnchangedP(unsigned int size, void* x, void* pt);
//#ifdef SOFA_GPU_CUDA_DOUBLE
//void Test2DAdapterCuda3d_computeTriangleNormal(const void* x, const void* n);
//#endif
}// extern "C"

typedef unsigned int Index;

// NOTE: must be equivalent to the Test2DAdapterData::TriangleData
template <class Real>
struct TriangleData {
    Index nodes[3];
    CudaVec3<Real> normal;
    Real functional;
    CudaVec3<Real> gradient[3];
};

// NOTE: must be equivalent to the Test2DAdapterData::PointData
template <class Real>
struct PointData {
    bool bBoundary;

    unsigned int nNeighboursPt;
    const Index *neighboursPt;

    unsigned int nNeighboursTri;
    const Index *neighboursTri;

    bool bAccepted; /// New position has been accepted in current step.
    CudaVec3<Real> oldpos;
    Real oldworst;
    Real newworst;

    Index mintri;
    CudaVec3<Real> grad;
};

//////////////////////
// GPU-side methods //
//////////////////////

template<class Real>
__device__ CudaVec3<Real> computeTriangleNormal(const CudaVec3<Real>* x,
    const Index nodes[3])
{
    CudaVec3<Real> A, B;
    A = x[ nodes[1] ] - x[ nodes[0] ];
    B = x[ nodes[2] ] - x[ nodes[0] ];

    CudaVec3<Real> normal = CudaVec3<Real>::make(0.0, 0.0, 0.0);

    Real An = invnorm(A), Bn = invnorm(B);
    if (An > 1e-20 && Bn > 1e-20) {
        A = A*An;
        B = B*Bn;
        normal = cross(A, B);
        normal = normal * invnorm(normal);
    }

    return normal;
}

template<class Real>
__device__ Real getMinFunc(Index v, const TriangleData<Real>* tri,
    const PointData<Real>* pt)
{
    unsigned int nElem = pt[v].nNeighboursTri;
    Real value = 1.0;
    // TODO: do some unrolling?
    for (Index it=0; it<nElem; it++) {
        if (value > tri[ pt[v].neighboursTri[it] ].functional) {
            value = tri[ pt[v].neighboursTri[it] ].functional;
        }
    }

    return value;
}

template<class Real>
__device__ Real functionalGeom(const Index t,
    const CudaVec3<Real>* x, const TriangleData<Real>* tri)
{
    // TODO: move outside, pass nodes as arguments

    // TODO: we can precompute these, is it worth it?
    CudaVec3<Real> ab = x[ tri[t].nodes[1] ] - x[ tri[t].nodes[0] ];
    CudaVec3<Real> ca = x[ tri[t].nodes[0] ] - x[ tri[t].nodes[2] ];
    CudaVec3<Real> cb = x[ tri[t].nodes[1] ] - x[ tri[t].nodes[2] ];

    // Normalizing factor so that the value is 1 in maximum
    // TODO: does compiler precompute this? NOTE: is float
    Real m = 2 * sqrt(3.0f);

    m *= norm(cross(ca,cb)); // || CA × CB ||
    m /= norm2(ca) + norm2(ab) + norm2(cb);

    // Is triangle inverted?
    CudaVec3<Real> nnew = computeTriangleNormal<Real>(x, tri[t].nodes);
    if (dot(nnew, tri[t].normal) < 0.0) {
        m *= -1.0;
    }

    return m;
}

template<class Real>
__device__ __inline__ Index translateIndexInTriangle(Index index,
    const TriangleData<Real> &tri)
{
    if (tri.nodes[0] == index) return 0;
    if (tri.nodes[1] == index) return 1;
    return 2;
}

//////////////////////
// Kernels          //
//////////////////////

template<class Real>
__global__ void Test2DAdapterCuda3t_computeTriangleNormal_kernel(unsigned int size,
    const CudaVec3<Real>* x, TriangleData<Real>* tri)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {
        tri[index].normal = computeTriangleNormal(x, tri[index].nodes);
    }
}

template<class Real>
__global__ void Test2DAdapterCuda3t_functionalGeom_kernel(unsigned int size,
    const CudaVec3<Real>* x, TriangleData<Real>* tri)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {
        tri[index].functional = functionalGeom(index, x, tri);
    }
}


// Laplacian smoothing
template<class Real>
__global__ void Test2DAdapterCuda3t_smoothLaplacian_kernel(unsigned int size,
    CudaVec3<Real>* x, const TriangleData<Real>* tri, PointData<Real>* pt,
    const Index *indices)
{
    typedef CudaVec3<Real> Vec3;

    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {

        Index v = indices[index];

        pt[v].oldpos = x[v];
        pt[v].oldworst = getMinFunc(v, tri, pt);
        pt[v].bAccepted = false;

        // Compute centroid of polygon from 1-ring around the vertex
        Vec3 xnew = Vec3::make(0.0, 0.0, 0.0);
        for (Index ie=0; ie<pt[v].nNeighboursPt; ie++) {
            xnew += x[ pt[v].neighboursPt[ie] ];
        }
        x[v] = xnew / Real(pt[v].nNeighboursPt);
    }
}

// Search for maximum of the functional
template<class Real>
__global__ void Test2DAdapterCuda3t_smoothOptimize_kernel(unsigned int size,
    CudaVec3<Real>* x, const TriangleData<Real>* tri, PointData<Real>* pt,
    const Index *indices)
{
    typedef CudaVec3<Real> Vec3;

    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {

        Index v = indices[index];
        Vec3 xold = x[v];

        pt[v].oldpos = x[v];
        pt[v].oldworst = getMinFunc(v, tri, pt);
        pt[v].bAccepted = false;

        unsigned int nElem = pt[v].nNeighboursTri;

        // Compute gradients
        // TODO: do it once for all elements!
        Vec3 grad[10]; // TODO: Vec3 grad[nElem];
        if (nElem > 10) nElem = 10; // XXX
        Real delta = 1e-5;

        // NOTE: Constrained to 2D!
        // TODO: can we use shared memory here?
        // -- X
        x[v].x += delta;
        for (Index it=0; it<nElem; it++) {
            Real m = functionalGeom<Real>(pt[v].neighboursTri[it], x, tri);
            grad[it].x = (m - tri[ pt[v].neighboursTri[it] ].functional)/delta;
        }
        // -- Y
        x[v].x = xold.x;
        x[v].y += delta;
        for (Index it=0; it<nElem; it++) {
            Real m = functionalGeom<Real>(pt[v].neighboursTri[it], x, tri);
            grad[it].y = (m - tri[ pt[v].neighboursTri[it] ].functional)/delta;
        }

        // Find smallest functional with non-zero gradient
        Index imin = 0;
        Real fmin = 1.0;
        for (Index it=0; it<nElem; it++) {
            if ((tri[ pt[v].neighboursTri[it] ].functional < fmin) &&
                (norm2(grad[it]) > 1e-15)) {
                fmin = tri[ pt[v].neighboursTri[it] ].functional;
                imin = it;
            }
        }

        Vec3 step = grad[imin];
        // Find out step size
        Real gamma = 0.05;
        //gamma *= step.norm();
        step = step * invnorm(step);

        x[v] = xold + gamma*step;
    }
}

template<class Real>
__global__ void Test2DAdapterCuda3t_reduceStep_kernel(unsigned int size,
    CudaVec3<Real>* x, const PointData<Real>* pt, const Index *indices)
{
    typedef CudaVec3<Real> Vec3;

    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {

        Index v = indices[index];
        if (!pt[v].bAccepted) {
            // The correct step size is best found empiricaly
            x[v] = pt[v].oldpos + (x[v] - pt[v].oldpos) * Real(2.0/3.0);
            //x[v] = (x[v] + pt[v].oldpos)/2.0;
        }
    }
}

template<class Real>
__global__ void Test2DAdapterCuda3t_restoreUnchanged_kernel(unsigned int size,
    CudaVec3<Real>* x, const PointData<Real>* pt, const Index *indices)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {
        Index v = indices[index];
        if (!pt[v].bAccepted) {
            x[v] = pt[v].oldpos;
        }
    }
}

template<class Real>
__global__ void Test2DAdapterCuda3t_testAcceptable_kernel(unsigned int size,
    CudaVec3<Real>* x, const TriangleData<Real>* tri, PointData<Real>* pt, const Index *indices, float tolerance)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {

        //// This check is not worth the effort
        //if ((xold - x[v]).norm2() < 1e-8) {
        //    // No change in position
        //    //std::cout << "No change in position for " << v << "\n";
        //    break;
        //}

        Index v = indices[index];
        //if (!pt[v].bAccepted) { // TODO

        // We accept any change that doesn't decrease worst metric for the
        // triangle set.
        Real newworst = getMinFunc(v, tri, pt);
        if (newworst >= (pt[v].oldworst + tolerance)) {
            pt[v].bAccepted = true;
        }
        pt[v].newworst = newworst;

        //}
    }
}

///// parallel

template<class Real>
__global__ void Test2DAdapterCuda3t_prepareGradients_kernel(unsigned int size,
    CudaVec3<Real>* x, TriangleData<Real>* tri)
{
    typedef CudaVec3<Real> Vec3;

    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {
        // TODO: handle boundary vertices

        for (int i=0; i<3; i++) {
            // For each corner

            Real delta = 1e-5;

            Index v = tri[index].nodes[i];
            Vec3 xold = x[v];

            // NOTE: Constrained to 2D!
            // TODO: can we use shared memory here?
            // -- X
            x[v].x += delta;
            Real m = functionalGeom<Real>(index, x, tri);
            tri[index].gradient[i].x = (m - tri[index].functional)/delta;
            // -- Y
            x[v].x = xold.x;
            x[v].y += delta;
            m = functionalGeom<Real>(index, x, tri);
            tri[index].gradient[i].y = (m - tri[index].functional)/delta;

            x[v].y = xold.y;
        }
    }
}

template<class Real>
__global__ void Test2DAdapterCuda3t_smoothParallel_kernel(unsigned int size,
    CudaVec3<Real>* x, const TriangleData<Real>* tri, PointData<Real>* pt)
{
    typedef CudaVec3<Real> Vec3;

    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {

        pt[index].oldpos = x[index];
        pt[index].oldworst = getMinFunc(index, tri, pt);
        pt[index].bAccepted = false;

        unsigned int nElem = pt[index].nNeighboursTri;

        // Find smallest functional with non-zero gradient
        Index tmin = Index(-1);
        Real fmin = 1.0;
        Vec3 step = Vec3::make(0.0, 0.0, 0.0);
        if (!pt[index].bBoundary) {
        for (Index it=0; it<nElem; it++) {
            Index t = pt[index].neighboursTri[it];
            Vec3 tg = tri[t].gradient[ translateIndexInTriangle(index, tri[t]) ];
            if ((tri[t].functional < fmin) && (norm2(tg) > 1e-15)) {
                fmin = tri[t].functional;
                tmin = t;
                step = tg;
            }
        }
        }
        pt[index].mintri = tmin;
        pt[index].grad = step;

        // Sync -- we need mintri to be available for all points
        __syncthreads();

        // Consult neighbourhood and make an estimate
        Vec3 ns = Vec3::make(0.0, 0.0, 0.0);
        if (!pt[index].bBoundary) {

        for (Index it=0; it<nElem; it++) {
            Index t = pt[index].neighboursTri[it];

            Index otherp[2];
            Index othert[2];
            for (int v=0, i=0; v<3; v++) {
                if (tri[t].nodes[v] == index) continue;
                otherp[i] = tri[t].nodes[v];
                othert[i] = pt[ otherp[i] ].mintri;
                i++;
            }

            Vec3 tmp = step;
            if (tmin == othert[0]) tmp = tmp / Real(2.0);
            if (tmin == othert[1]) tmp = tmp / Real(2.0);

            if (tmin != othert[0] && tmin != othert[1]) {
                tmp += (pt[ otherp[0] ].grad + pt[ otherp[1] ].grad)/Real(2.0);
            } else if (tmin != othert[0]) {
                tmp += pt[ otherp[0] ].grad;
            } else if (tmin != othert[1]) {
                tmp += pt[ otherp[1] ].grad;
            } 

            ns += tmp;
        }
        ns = ns / Real(nElem);

        }

        Real gamma = 0.005;

        x[index] += gamma*ns;
    }
}

template<class Real>
__global__ void Test2DAdapterCuda3t_reduceStepP_kernel(unsigned int size,
    CudaVec3<Real>* x, const PointData<Real>* pt)
{
    typedef CudaVec3<Real> Vec3;

    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {

        if (!pt[index].bAccepted) {
            // The correct step size is best found empiricaly
            x[index] = pt[index].oldpos
                + (x[index] - pt[index].oldpos) * Real(2.0/3.0);
            //x[index] = (x[v] + pt[index].oldpos)/2.0;
        }
    }
}

template<class Real>
__global__ void Test2DAdapterCuda3t_testAcceptableP_kernel(unsigned int size,
    CudaVec3<Real>* x, const TriangleData<Real>* tri, PointData<Real>* pt,
    float tolerance)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {

        //// This check is not worth the effort
        //if ((xold - x[index]).norm2() < 1e-8) {
        //    // No change in position
        //    //std::cout << "No change in position for " << index << "\n";
        //    break;
        //}

        //if (!pt[index].bAccepted) { // TODO

        // We accept any change that doesn't decrease worst metric for the
        // triangle set.
        Real newworst = getMinFunc(index, tri, pt);
        if (newworst >= (pt[index].oldworst + tolerance)) {
            pt[index].bAccepted = true;
        }
        pt[index].newworst = newworst;

        //}
    }
}

template<class Real>
__global__ void Test2DAdapterCuda3t_restoreUnchangedP_kernel(unsigned int size,
    CudaVec3<Real>* x, const PointData<Real>* pt)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {
        if (!pt[index].bAccepted) {
            x[index] = pt[index].oldpos;
        }
    }
}



//////////////////////
// CPU-side methods //
//////////////////////


void Test2DAdapterCuda3f_computeTriangleNormal(unsigned int size, const void* x, void* tri)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    Test2DAdapterCuda3t_computeTriangleNormal_kernel<float><<< grid, threads >>>(size, (const CudaVec3<float>*)x, (TriangleData<float>*)tri);
    mycudaDebugError("Test2DAdapterCuda3t_computeTriangleNormal_kernel<float>");
}

void Test2DAdapterCuda3f_functionalGeom(unsigned int size, const void* x, void* tri)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    Test2DAdapterCuda3t_functionalGeom_kernel<float><<< grid, threads >>>(size, (const CudaVec3<float>*)x, (TriangleData<float>*)tri);
    mycudaDebugError("Test2DAdapterCuda3t_functionalGeom_kernel<float>");
}

void Test2DAdapterCuda3f_reduceStep(unsigned int size, void* x, void* pt, void* indices)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    Test2DAdapterCuda3t_reduceStep_kernel<float><<< grid, threads >>>(size, (CudaVec3<float>*)x, (const PointData<float>*)pt, (const Index*) indices);
    mycudaDebugError("Test2DAdapterCuda3t_reduceStep_kernel<float>");
}

void Test2DAdapterCuda3f_restoreUnchanged(unsigned int size, void* x, void* pt, void* indices)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    Test2DAdapterCuda3t_restoreUnchanged_kernel<float><<< grid, threads >>>(size, (CudaVec3<float>*)x, (const PointData<float>*)pt, (const Index*) indices);
    mycudaDebugError("Test2DAdapterCuda3t_restoreUnchanged_kernel<float>");
}

void Test2DAdapterCuda3f_smooth(unsigned int size, void* x, void* tri, void* pt, void* indices)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    //Test2DAdapterCuda3t_smoothLaplacian_kernel<float><<< grid, threads >>>(
    //    size, (CudaVec3<float>*)x, (const TriangleData<float>*)tri,
    //    (PointData<float>*)pt, (const Index*) indices);
    //mycudaDebugError("Test2DAdapterCuda3t_smoothLaplacian_kernel<float>");
    Test2DAdapterCuda3t_smoothOptimize_kernel<float><<< grid, threads >>>(
        size, (CudaVec3<float>*)x, (const TriangleData<float>*)tri,
        (PointData<float>*)pt, (const Index*) indices);
    mycudaDebugError("Test2DAdapterCuda3t_smoothOptimize_kernel<float>");
}

void Test2DAdapterCuda3f_testAcceptable(unsigned int size, void* x, const void* tri, void* pt, void* indices, float tolerance)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    Test2DAdapterCuda3t_testAcceptable_kernel<float><<< grid, threads >>>(size, (CudaVec3<float>*)x, (const TriangleData<float>*)tri, (PointData<float>*)pt, (const Index*) indices, tolerance);
    mycudaDebugError("Test2DAdapterCuda3t_testAcceptable_kernel<float>");
}


/// Specific to parallel version

void Test2DAdapterCuda3f_prepareGradients(unsigned int size, const void* x,
    void* tri)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    Test2DAdapterCuda3t_prepareGradients_kernel<float>
        <<< grid, threads >>>(
            size, (CudaVec3<float>*)x, (TriangleData<float>*)tri);
    mycudaDebugError("Test2DAdapterCuda3t_prepareGradients_kernel<float>");
}

void Test2DAdapterCuda3f_smoothParallel(unsigned int size, void* x,
    const void* tri, void* pt)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    Test2DAdapterCuda3t_smoothParallel_kernel<float>
        <<< grid, threads >>>(
            size, (CudaVec3<float>*)x, (const TriangleData<float>*)tri,
            (PointData<float>*)pt);
    mycudaDebugError("Test2DAdapterCuda3t_smoothParallel_kernel<float>");
}

void Test2DAdapterCuda3f_reduceStepP(unsigned int size, void* x, void* pt)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    Test2DAdapterCuda3t_reduceStepP_kernel<float>
        <<< grid, threads >>>(
            size, (CudaVec3<float>*)x, (const PointData<float>*)pt);
    mycudaDebugError("Test2DAdapterCuda3t_reduceStepP_kernel<float>");
}

void Test2DAdapterCuda3f_testAcceptableP(unsigned int size, void* x,
    const void* tri, void* pt, float tolerance)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    Test2DAdapterCuda3t_testAcceptableP_kernel<float>
        <<< grid, threads >>>(
            size, (CudaVec3<float>*)x, (const TriangleData<float>*)tri,
            (PointData<float>*)pt, tolerance);
    mycudaDebugError("Test2DAdapterCuda3t_testAcceptableP_kernel<float>");
}

void Test2DAdapterCuda3f_restoreUnchangedP(unsigned int size, void* x,
    void* pt)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    Test2DAdapterCuda3t_restoreUnchangedP_kernel<float>
        <<< grid, threads >>>(
            size, (CudaVec3<float>*)x, (const PointData<float>*)pt);
    mycudaDebugError("Test2DAdapterCuda3t_restoreUnchangedP_kernel<float>");
}


#if defined(__cplusplus) && CUDA_VERSION < 2000
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
