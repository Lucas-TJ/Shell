#include "hip/hip_runtime.h"
#include <sofa/helper/fixed_array.h>
#include <sofa/gpu/cuda/CudaCommon.h>
#include <sofa/gpu/cuda/CudaMath.h>
//#include <hip/hip_runtime.h>

#if defined(__cplusplus) && CUDA_VERSION < 2000
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

extern "C"
{
void Test2DAdapterCuda3f_computeTriangleNormal(unsigned int size, const void* x, void* tri);
void Test2DAdapterCuda3f_functionalGeom(unsigned int size, const void* x, void* tri);
void Test2DAdapterCuda3f_reduceStep(unsigned int size, void* x, void* pt, void* indices);
void Test2DAdapterCuda3f_restoreUnchanged(unsigned int size, void* x, void* pt, void* indices);
void Test2DAdapterCuda3f_smooth(unsigned int size, void* x, void* tri, void* pt, void* indices);
void Test2DAdapterCuda3f_testAcceptable(unsigned int size, void* x, const void* tri, void* pt, void* indices, float tolerance);
//#ifdef SOFA_GPU_CUDA_DOUBLE
//void Test2DAdapterCuda3d_computeTriangleNormal(const void* x, const void* n);
//#endif
}// extern "C"

typedef unsigned int Index;

// NOTE: should be equivalent to the Test2DAdapterData::TriangleData
template <class Real>
struct TriangleData {
    Index nodes[3];
    CudaVec3<Real> normal;
    Real functional;
};

template <class Real>
struct PointData {
    unsigned int nNeighboursPt;
    const Index *neighboursPt;

    unsigned int nNeighboursTri;
    const Index *neighboursTri;

    bool bAccepted; /// New position has been accepted in current step.
    CudaVec3<Real> oldpos;
    Real oldworst;
    Real newworst;
};

//////////////////////
// GPU-side methods //
//////////////////////

template<class Real>
__device__ CudaVec3<Real> computeTriangleNormal(const CudaVec3<Real>* x,
    const Index nodes[3])
{
    CudaVec3<Real> A, B;
    A = x[ nodes[1] ] - x[ nodes[0] ];
    B = x[ nodes[2] ] - x[ nodes[0] ];

    CudaVec3<Real> normal = CudaVec3<Real>::make(0.0, 0.0, 0.0);

    Real An = invnorm(A), Bn = invnorm(B);
    if (An > 1e-20 && Bn > 1e-20) {
        A = A*An;
        B = B*Bn;
        normal = cross(A, B);
        normal = normal * invnorm(normal);
    }

    return normal;
}

template<class Real>
__device__ Real getMinFunc(Index v, const TriangleData<Real>* tri,
    const PointData<Real>* pt)
{
    unsigned int nElem = pt[v].nNeighboursTri;
    Real value = 1.0;
    // TODO: do some unrolling?
    for (Index it=0; it<nElem; it++) {
        if (value > tri[ pt[v].neighboursTri[it] ].functional) {
            value = tri[ pt[v].neighboursTri[it] ].functional;
        }
    }

    return value;
}

template<class Real>
__device__ Real functionalGeom(const Index t,
    const CudaVec3<Real>* x, const TriangleData<Real>* tri)
{
    // TODO: move outside, pass nodes as arguments

    // TODO: we can precompute these, is it worth it?
    CudaVec3<Real> ab = x[ tri[t].nodes[1] ] - x[ tri[t].nodes[0] ];
    CudaVec3<Real> ca = x[ tri[t].nodes[0] ] - x[ tri[t].nodes[2] ];
    CudaVec3<Real> cb = x[ tri[t].nodes[1] ] - x[ tri[t].nodes[2] ];

    // Normalizing factor so that the value is 1 in maximum
    // TODO: does compiler precompute this? NOTE: is float
    Real m = 2 * sqrt(3.0f);

    m *= norm(cross(ca,cb)); // || CA × CB ||
    m /= norm2(ca) + norm2(ab) + norm2(cb);

    // Is triangle inverted?
    CudaVec3<Real> nnew = computeTriangleNormal<Real>(x, tri[t].nodes);
    if (dot(nnew, tri[t].normal) < 0.0) {
        m *= -1.0;
    }

    return m;
}

//////////////////////
// Kernels          //
//////////////////////

template<class Real>
__global__ void Test2DAdapterCuda3t_computeTriangleNormal_kernel(unsigned int size,
    const CudaVec3<Real>* x, TriangleData<Real>* tri)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {
        tri[index].normal = computeTriangleNormal(x, tri[index].nodes);
    }
}

template<class Real>
__global__ void Test2DAdapterCuda3t_functionalGeom_kernel(unsigned int size,
    const CudaVec3<Real>* x, TriangleData<Real>* tri)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {
        tri[index].functional = functionalGeom(index, x, tri);
    }
}


// Laplacian smoothing
template<class Real>
__global__ void Test2DAdapterCuda3t_smoothLaplacian_kernel(unsigned int size,
    CudaVec3<Real>* x, const TriangleData<Real>* tri, PointData<Real>* pt,
    const Index *indices)
{
    typedef CudaVec3<Real> Vec3;

    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {

        Index v = indices[index];

        pt[v].oldpos = x[v];
        pt[v].oldworst = getMinFunc(v, tri, pt);
        pt[v].bAccepted = false;

        // Compute centroid of polygon from 1-ring around the vertex
        Vec3 xnew = Vec3::make(0.0, 0.0, 0.0);
        for (Index ie=0; ie<pt[v].nNeighboursPt; ie++) {
            xnew += x[ pt[v].neighboursPt[ie] ];
        }
        x[v] = xnew / Real(pt[v].nNeighboursPt);
    }
}

// Search for maximum of the functional
template<class Real>
__global__ void Test2DAdapterCuda3t_smoothOptimize_kernel(unsigned int size,
    CudaVec3<Real>* x, const TriangleData<Real>* tri, PointData<Real>* pt,
    const Index *indices)
{
    typedef CudaVec3<Real> Vec3;

    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {

        Index v = indices[index];
        Vec3 xold = x[v];

        pt[v].oldpos = x[v];
        pt[v].oldworst = getMinFunc(v, tri, pt);
        pt[v].bAccepted = false;

        unsigned int nElem = pt[v].nNeighboursTri;

        // Compute gradients
        // TODO: do it once for all elements!
        Vec3 grad[10]; // TODO: Vec3 grad[nElem];
        if (nElem > 10) nElem = 10; // XXX
        Real delta = 1e-5;

        // NOTE: Constrained to 2D!
        // TODO: can we use shared memory here?
        // -- X
        x[v].x += delta;
        for (Index it=0; it<nElem; it++) {
            Real m = functionalGeom<Real>(pt[v].neighboursTri[it], x, tri);
            grad[it].x = (m - tri[ pt[v].neighboursTri[it] ].functional)/delta;
        }
        // -- Y
        x[v].x = xold.x;
        x[v].y += delta;
        for (Index it=0; it<nElem; it++) {
            Real m = functionalGeom<Real>(pt[v].neighboursTri[it], x, tri);
            grad[it].y = (m - tri[ pt[v].neighboursTri[it] ].functional)/delta;
        }

        // Find smallest functional with non-zero gradient
        Index imin = 0;
        Real fmin = 1.0;
        for (Index it=0; it<nElem; it++) {
            if ((tri[ pt[v].neighboursTri[it] ].functional < fmin) &&
                (norm2(grad[it]) > 1e-15)) {
                fmin = tri[ pt[v].neighboursTri[it] ].functional;
                imin = it;
            }
        }

        Vec3 step = grad[imin];
        // Find out step size
        Real gamma = 0.05;
        //gamma *= step.norm();
        step = step * invnorm(step);

        x[v] = xold + gamma*step;
    }
}

template<class Real>
__global__ void Test2DAdapterCuda3t_reduceStep_kernel(unsigned int size,
    CudaVec3<Real>* x, const PointData<Real>* pt, const Index *indices)
{
    typedef CudaVec3<Real> Vec3;

    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {

        Index v = indices[index];
        if (!pt[v].bAccepted) {
            // The correct step size is best found empiricaly
            x[v] = pt[v].oldpos + (x[v] - pt[v].oldpos) * Real(2.0/3.0);
            //x[v] = (x[v] + pt[v].oldpos)/2.0;
        }
    }
}

template<class Real>
__global__ void Test2DAdapterCuda3t_restoreUnchanged_kernel(unsigned int size,
    CudaVec3<Real>* x, const PointData<Real>* pt, const Index *indices)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {

        Index v = indices[index];
        if (!pt[v].bAccepted) {
            x[v] = pt[v].oldpos;
        }
    }
}

template<class Real>
__global__ void Test2DAdapterCuda3t_testAcceptable_kernel(unsigned int size,
    CudaVec3<Real>* x, const TriangleData<Real>* tri, PointData<Real>* pt, const Index *indices, float tolerance)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {

        //// This check is not worth the effort
        //if ((xold - x[v]).norm2() < 1e-8) {
        //    // No change in position
        //    //std::cout << "No change in position for " << v << "\n";
        //    break;
        //}

        Index v = indices[index];
        //if (!pt[v].bAccepted) { // TODO

        // We accept any change that doesn't decrease worst metric for the
        // triangle set.
        Real newworst = getMinFunc(v, tri, pt);
        if (newworst >= (pt[v].oldworst + tolerance)) {
            pt[v].bAccepted = true;
        }
        pt[v].newworst = newworst;

        //}
    }
}


//////////////////////
// CPU-side methods //
//////////////////////


void Test2DAdapterCuda3f_computeTriangleNormal(unsigned int size, const void* x, void* tri)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    Test2DAdapterCuda3t_computeTriangleNormal_kernel<float><<< grid, threads >>>(size, (const CudaVec3<float>*)x, (TriangleData<float>*)tri);
    mycudaDebugError("Test2DAdapterCuda3t_computeTriangleNormal_kernel<float>");
}

void Test2DAdapterCuda3f_functionalGeom(unsigned int size, const void* x, void* tri)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    Test2DAdapterCuda3t_functionalGeom_kernel<float><<< grid, threads >>>(size, (const CudaVec3<float>*)x, (TriangleData<float>*)tri);
    mycudaDebugError("Test2DAdapterCuda3t_functionalGeom_kernel<float>");
}

void Test2DAdapterCuda3f_reduceStep(unsigned int size, void* x, void* pt, void* indices)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    Test2DAdapterCuda3t_reduceStep_kernel<float><<< grid, threads >>>(size, (CudaVec3<float>*)x, (const PointData<float>*)pt, (const Index*) indices);
    mycudaDebugError("Test2DAdapterCuda3t_reduceStep_kernel<float>");
}

void Test2DAdapterCuda3f_restoreUnchanged(unsigned int size, void* x, void* pt, void* indices)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    Test2DAdapterCuda3t_restoreUnchanged_kernel<float><<< grid, threads >>>(size, (CudaVec3<float>*)x, (const PointData<float>*)pt, (const Index*) indices);
    mycudaDebugError("Test2DAdapterCuda3t_restoreUnchanged_kernel<float>");
}

void Test2DAdapterCuda3f_smooth(unsigned int size, void* x, void* tri, void* pt, void* indices)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    //Test2DAdapterCuda3t_smoothLaplacian_kernel<float><<< grid, threads >>>(
    //    size, (CudaVec3<float>*)x, (const TriangleData<float>*)tri,
    //    (PointData<float>*)pt, (const Index*) indices);
    //mycudaDebugError("Test2DAdapterCuda3t_smoothLaplacian_kernel<float>");
    Test2DAdapterCuda3t_smoothOptimize_kernel<float><<< grid, threads >>>(
        size, (CudaVec3<float>*)x, (const TriangleData<float>*)tri,
        (PointData<float>*)pt, (const Index*) indices);
    mycudaDebugError("Test2DAdapterCuda3t_smoothOptimize_kernel<float>");
}

void Test2DAdapterCuda3f_testAcceptable(unsigned int size, void* x, const void* tri, void* pt, void* indices, float tolerance)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    Test2DAdapterCuda3t_testAcceptable_kernel<float><<< grid, threads >>>(size, (CudaVec3<float>*)x, (const TriangleData<float>*)tri, (PointData<float>*)pt, (const Index*) indices, tolerance);
    mycudaDebugError("Test2DAdapterCuda3t_testAcceptable_kernel<float>");
}



#if defined(__cplusplus) && CUDA_VERSION < 2000
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
