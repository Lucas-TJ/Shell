#include "hip/hip_runtime.h"
#include <sofa/helper/fixed_array.h>
#include <sofa/gpu/cuda/CudaCommon.h>
#include <sofa/gpu/cuda/CudaMath.h>
//#include <hip/hip_runtime.h>

#if defined(__cplusplus) && CUDA_VERSION < 2000
namespace sofa
{
namespace gpu
{
namespace cuda
{
#endif

extern "C"
{
void Test2DAdapterCuda3f_computeTriangleNormal(unsigned int size, const void* x, void* tri);
void Test2DAdapterCuda3f_functionalGeom(unsigned int size, const void* x, void* tri);
void Test2DAdapterCuda3f_reduceStep(unsigned int size, void* x, void* pt, void* indices);
void Test2DAdapterCuda3f_restoreUnchanged(unsigned int size, void* x, void* pt, void* indices);
void Test2DAdapterCuda3f_smooth(unsigned int size, void* x, void* tri, void* pt, void* indices);
void Test2DAdapterCuda3f_testAcceptable(unsigned int size, void* x, const void* tri, void* pt, void* indices, float tolerance);
//#ifdef SOFA_GPU_CUDA_DOUBLE
//void Test2DAdapterCuda3d_computeTriangleNormal(const void* x, const void* n);
//#endif
}// extern "C"

typedef unsigned int Index;

// NOTE: should be equivalent to the Test2DAdapterData::TriangleData
template <class Real>
struct TriangleData {
    Index nodes[3];
    CudaVec3<Real> normal;
    Real functional;
};

template <class Real>
struct PointData {
    unsigned int nNeighboursPt;
    const Index *neighboursPt;

    unsigned int nNeighboursTri;
    const Index *neighboursTri;

    bool bAccepted; /// New position has been accepted in current step.
    CudaVec3<Real> oldpos;
    Real oldworst;
    Real newworst;
};

//////////////////////
// GPU-side methods //
//////////////////////

template<class Real>
__device__ CudaVec3<Real> computeTriangleNormal(const CudaVec3<Real>* x,
    const Index nodes[3])
{
    CudaVec3<Real> A, B;
    A = x[ nodes[1] ] - x[ nodes[0] ];
    B = x[ nodes[2] ] - x[ nodes[0] ];

    CudaVec3<Real> normal = CudaVec3<Real>::make(0.0, 0.0, 0.0);

    Real An = invnorm(A), Bn = invnorm(B);
    if (An > 1e-20 && Bn > 1e-20) {
        A = A*An;
        B = B*Bn;
        normal = cross(A, B);
        normal = normal * invnorm(normal);
    }

    return normal;
}

template<class Real>
__device__ Real getMinFunc(Index v, const TriangleData<Real>* tri,
    const PointData<Real>* pt)
{
    unsigned int nElem = pt[v].nNeighboursTri;
    Real value = 1.0;
    // TODO: do some unrolling?
    for (Index it=0; it<nElem; it++) {
        if (value > tri[ pt[v].neighboursTri[it] ].functional) {
            value = tri[ pt[v].neighboursTri[it] ].functional;
        }
    }

    return value;
}

//////////////////////
// Kernels          //
//////////////////////

template<class Real>
__global__ void Test2DAdapterCuda3t_computeTriangleNormal_kernel(unsigned int size,
    const CudaVec3<Real>* x, TriangleData<Real>* tri)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {
        tri[index].normal = computeTriangleNormal(x, tri[index].nodes);
    }
}

template<class Real>
__global__ void Test2DAdapterCuda3t_functionalGeom_kernel(unsigned int size,
    const CudaVec3<Real>* x, TriangleData<Real>* tri)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {
        // TODO: we can precompute these, is it worth it?
        CudaVec3<Real> ab = x[ tri[index].nodes[1] ] - x[ tri[index].nodes[0] ];
        CudaVec3<Real> ca = x[ tri[index].nodes[0] ] - x[ tri[index].nodes[2] ];
        CudaVec3<Real> cb = x[ tri[index].nodes[1] ] - x[ tri[index].nodes[2] ];

        // Normalizing factor so that the value is 1 in maximum
        Real m = 2 * sqrt(3.0f); // TODO: does compiler precompute this? NOTE: is float
        m *= norm(cross(ca,cb)); // || CA × CB ||
        m /= norm2(ca) + norm2(ab) + norm2(cb);

        // Is triangle inverted?
        CudaVec3<Real> nnew = computeTriangleNormal<Real>(x, tri[index].nodes);
        if (dot(nnew, tri[index].normal) < 0.0) {
            m *= -1.0;
        }

        tri[index].functional = m;
    }
}


template<class Real>
__global__ void Test2DAdapterCuda3t_smoothLaplacian_kernel(unsigned int size,
    CudaVec3<Real>* x, const TriangleData<Real>* tri, PointData<Real>* pt,
    const Index *indices)
{
    typedef CudaVec3<Real> Vec3;

    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {

        Index v = indices[index];

        pt[v].oldpos = x[v];
        pt[v].oldworst = getMinFunc(v, tri, pt);

        // Compute centroid of polygon from 1-ring around the vertex
        Vec3 xnew = Vec3::make(0.0, 0.0, 0.0);
        for (Index ie=0; ie<pt[v].nNeighboursPt; ie++) {
            xnew += x[ pt[v].neighboursPt[ie] ];
        }
        x[v] = xnew / Real(pt[v].nNeighboursPt);

        pt[v].bAccepted = false;
    }
}

template<class Real>
__global__ void Test2DAdapterCuda3t_reduceStep_kernel(unsigned int size,
    CudaVec3<Real>* x, const PointData<Real>* pt, const Index *indices)
{
    typedef CudaVec3<Real> Vec3;

    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {

        Index v = indices[index];
        if (!pt[v].bAccepted) {
            // The correct step size is best found empiricaly
            x[v] = (x[v] + pt[v].oldpos) * Real(2.0/3.0);
            //x[v] = (x[v] + pt[v].oldpos)/2.0;
        }
    }
}

template<class Real>
__global__ void Test2DAdapterCuda3t_restoreUnchanged_kernel(unsigned int size,
    CudaVec3<Real>* x, const PointData<Real>* pt, const Index *indices)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {

        Index v = indices[index];
        if (!pt[v].bAccepted) {
            x[v] = pt[v].oldpos;
        }
    }
}

template<class Real>
__global__ void Test2DAdapterCuda3t_testAcceptable_kernel(unsigned int size,
    CudaVec3<Real>* x, const TriangleData<Real>* tri, PointData<Real>* pt, const Index *indices, float tolerance)
{
    int index = umul24(blockIdx.x,BSIZE)+threadIdx.x;
    if (index < size) {

        //// This check is not worth the effort
        //if ((xold - x[v]).norm2() < 1e-8) {
        //    // No change in position
        //    //std::cout << "No change in position for " << v << "\n";
        //    break;
        //}

        Index v = indices[index];
        //if (!pt[v].bAccepted) { // TODO

        // We accept any change that doesn't decrease worst metric for the
        // triangle set.
        Real newworst = getMinFunc(v, tri, pt);
        if (newworst >= (pt[v].oldworst + tolerance)) {
            pt[v].bAccepted = true;
        }
        pt[v].newworst = newworst;

        //}
    }
}


//////////////////////
// CPU-side methods //
//////////////////////


void Test2DAdapterCuda3f_computeTriangleNormal(unsigned int size, const void* x, void* tri)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    Test2DAdapterCuda3t_computeTriangleNormal_kernel<float><<< grid, threads >>>(size, (const CudaVec3<float>*)x, (TriangleData<float>*)tri);
    mycudaDebugError("Test2DAdapterCuda3t_computeTriangleNormal_kernel<float>");
}

void Test2DAdapterCuda3f_functionalGeom(unsigned int size, const void* x, void* tri)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    Test2DAdapterCuda3t_functionalGeom_kernel<float><<< grid, threads >>>(size, (const CudaVec3<float>*)x, (TriangleData<float>*)tri);
    mycudaDebugError("Test2DAdapterCuda3t_functionalGeom_kernel<float>");
}

void Test2DAdapterCuda3f_reduceStep(unsigned int size, void* x, void* pt, void* indices)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    Test2DAdapterCuda3t_reduceStep_kernel<float><<< grid, threads >>>(size, (CudaVec3<float>*)x, (const PointData<float>*)pt, (const Index*) indices);
    mycudaDebugError("Test2DAdapterCuda3t_reduceStep_kernel<float>");
}

void Test2DAdapterCuda3f_restoreUnchanged(unsigned int size, void* x, void* pt, void* indices)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    Test2DAdapterCuda3t_restoreUnchanged_kernel<float><<< grid, threads >>>(size, (CudaVec3<float>*)x, (const PointData<float>*)pt, (const Index*) indices);
    mycudaDebugError("Test2DAdapterCuda3t_restoreUnchanged_kernel<float>");
}

void Test2DAdapterCuda3f_smooth(unsigned int size, void* x, void* tri, void* pt, void* indices)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    Test2DAdapterCuda3t_smoothLaplacian_kernel<float><<< grid, threads >>>(size, (CudaVec3<float>*)x, (const TriangleData<float>*)tri, (PointData<float>*)pt, (const Index*) indices);
    mycudaDebugError("Test2DAdapterCuda3t_smoothLaplacian_kernel<float>");
}

void Test2DAdapterCuda3f_testAcceptable(unsigned int size, void* x, const void* tri, void* pt, void* indices, float tolerance)
{
    dim3 threads(BSIZE,1);
    dim3 grid((size+BSIZE-1)/BSIZE,1);
    Test2DAdapterCuda3t_testAcceptable_kernel<float><<< grid, threads >>>(size, (CudaVec3<float>*)x, (const TriangleData<float>*)tri, (PointData<float>*)pt, (const Index*) indices, tolerance);
    mycudaDebugError("Test2DAdapterCuda3t_testAcceptable_kernel<float>");
}



#if defined(__cplusplus) && CUDA_VERSION < 2000
} // namespace cuda
} // namespace gpu
} // namespace sofa
#endif
